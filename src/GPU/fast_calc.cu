#include "hip/hip_runtime.h"
#include <time.h>
#include <cstring>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <string>
using std::cin;
using std::cout;
using std::endl;
using std::ifstream;
using std::ios;
using std::ios_base;
using std::ofstream;

#include "hip/hip_runtime.h"
#include ""

#include "../header/define.hpp"
constexpr int DEBUG_SAMPLE_RATE_REV = 32;
int parallel_emit_sum = 1;    // 并行处理多个发射节点，优化使用

__device__ float dev_ele_coord_x[ELE_NO];    // 写到纹理内存里面
__device__ float dev_ele_coord_y[ELE_NO];    // 写到纹理内存里面
__device__ float dev_filter_data[OD];        // filter parameter

#include "fast_calc_kernel.cuh"
#include "helper_functions.cuh"

float image_data[PIC_RESOLUTION * PIC_RESOLUTION] = {0};
int image_point_count[PIC_RESOLUTION * PIC_RESOLUTION] = {0};

__global__ void calc_func(const int ele_emit_id, float *image_data,
                          int *point_count, const float *trans_sdata,
                          const int parallel_emit_sum) {
  int sound_speed = SOUND_SPEED;
  float fs = FS;
  float image_width = IMAGE_WIDTH;
  int point_length = DATA_DIAMETER / SOUND_SPEED * FS + 0.5;
  int middot =
    MIDDOT;    //发射前1us开始接收，也就是约为12.5个点之后发射,数据显示约16个点
  float tgc = TGC;
  float radius = RADIUS;
  float coord_step = COORD_STEP;
  int image_x_id = blockIdx.y;    //线
  int image_y_id = blockIdx.x;    //点
  int image_y_dim = gridDim.x;
  int recv_center_id = threadIdx.x;    // center of 接收阵元

  __shared__ float cache_image[2 * RCV_OFFSET];
  __shared__ int cache_point[2 * RCV_OFFSET];
  int cacheIndex = threadIdx.x;

  if (image_x_id < PIC_RESOLUTION && image_y_id < PIC_RESOLUTION &&
      recv_center_id < 2 * RCV_OFFSET) {
    // the above condition is bullshit

    float sum_image = 0;
    int sum_point = 0;
    float sample_coord_x = -image_width / 2 + coord_step * image_x_id;
    float sample_coord_y = -image_width / 2 + coord_step * image_y_id;

    for (int step_offset = 0; step_offset < parallel_emit_sum;
         step_offset += 1) {
      int step = ele_emit_id + step_offset;
      int send_id = step;                                     // as send_id
      int recv_id = send_id - RCV_OFFSET + recv_center_id;    //接收阵元
      recv_id = (recv_id + ELE_NO) % ELE_NO;
      float dis_snd =
        distance(dev_ele_coord_x[send_id], dev_ele_coord_y[send_id],
                 sample_coord_x, sample_coord_y);
      float dis_recv =
        distance(dev_ele_coord_x[recv_id], dev_ele_coord_y[recv_id],
                 sample_coord_x, sample_coord_y);
      float dis_origin = sqrtf(sample_coord_x * sample_coord_x +
                               sample_coord_y * sample_coord_y);

      // put dis_snd constraint onto for;
      // and since
      auto diff = send_id - recv_id;
      float recv_region = 244 * sqrtf(10 * dis_snd);
      bool is_valid = is_close(diff, recv_region);

      if (is_valid) {
        int waves = (dis_snd + dis_recv) / sound_speed * fs + 0.5;
        int magic = (waves + middot + (OD - 1 - 1) / 2);

        if ((magic > 100) && (magic <= point_length)) {
          // 2 * R * dis_snd * cosTheta
          // = R^2 + dis_snd^2 - |(x, y)|^2
          float angle = acosf(
            (radius * radius + dis_snd * dis_snd - dis_origin * dis_origin) /
            2 / radius / dis_snd);
          if ((angle < PI / 9)) {
            sum_image += trans_sdata[recv_id + magic * ELE_NO +
                                     step_offset * ELE_NO * NSAMPLE] *
                         expf(tgc * (waves - 1));
            sum_point += 1;
          }
        }
      }
    }
    cache_image[cacheIndex] = sum_image;
    cache_point[cacheIndex] = sum_point;

    __syncthreads();
    // sum up cache_image and cacheIndex, and i have way to make this part
    // disappear
    int step = blockDim.x / 2;
    while (step != 0) {
      if (cacheIndex < step) {
        cache_image[cacheIndex] += cache_image[cacheIndex + step];
        cache_point[cacheIndex] += cache_point[cacheIndex + step];
      }
      __syncthreads();
      step /= 2;
    }

    if (cacheIndex == 0) {
      int pixel_index = image_y_id + image_x_id * image_y_dim;    //线程块的索引
      image_data[pixel_index] = cache_image[0];
      point_count[pixel_index] = cache_point[0];
    }
  }
}

hipError_t precalcWithCuda(short *dev_data_samples_in_process, int ele_emit_id,
                            float *dev_sumdata, int *dev_sumpoint,
                            float *dev_filtered_data, float *dev_imagedata,
                            int *dev_pointcount, int parallel_emit_sum) {
  hipError_t cudaStatus;

  // kernel 1,kernel2 decode
  // kernel3 filter
  hipMemset(dev_filtered_data, 0,
             NSAMPLE * ELE_NO * sizeof(short) * parallel_emit_sum * 2);
  filter_func<<<4 * parallel_emit_sum, 512>>>(dev_filtered_data,
                                              dev_data_samples_in_process);
  // cudaStatus = hipDeviceSynchronize();

  // dim3 gridimage(PIC_RESOLUTION, PIC_RESOLUTION);
  // // dim3 threads(RCV_OFFSET);
  // calc_func<<<gridimage, 2 * RCV_OFFSET>>>(
  //   ele_emit_id, dev_imagedata, dev_pointcount, dev_filtered_data,
  //   parallel_emit_sum);    //启动一个二维的PIC_RESOLUTION*PIC_RESOLUTION个block，每个block里面RCV_OFFSET个thread





  fast_calc(dev_filtered_data, ele_emit_id, parallel_emit_sum, dev_sumdata, dev_sumpoint);

  // // Check for any errors launching the kernel
  cudaStatus = hipGetLastError();
  // if (cudaStatus != hipSuccess) {
  //   cout << "calcKernel launch failed: " << hipGetErrorString(cudaStatus);
  //   // goto Error;
  //   return cudaStatus;
  // }
  // // hipDeviceSynchronize();

  // //把所有的结果加到一起
  // add<<<32, 32>>>(dev_sumdata, dev_sumpoint, dev_imagedata, dev_pointcount);
  // cudaStatus = hipGetLastError();
  // if (cudaStatus != hipSuccess) {
  //   cout << "addKernel launch failed: " << hipGetErrorString(cudaStatus);
  //   // goto Error;
  //   return cudaStatus;
  // }

  return cudaStatus;
}



void write_txtfile(std::string output_path) {
  ofstream outfile(output_path);
  if (!outfile.is_open()) {
    cout << " the file open fail" << endl;
    exit(1);
  }

  for (int k = 0; k < PIC_RESOLUTION; k++) {
    for (int j = 0; j < PIC_RESOLUTION; j++) {
      if (image_point_count[k * PIC_RESOLUTION + j] == 0)
        outfile << image_data[k * PIC_RESOLUTION + j] << " ";
      else
        outfile << image_data[k * PIC_RESOLUTION + j] /
                     image_point_count[k * PIC_RESOLUTION + j]
                << " ";
    }
    outfile << "\r\n";
  }

  outfile.close();
}

int main(int argc, char const *argv[]) {
  time_t start, over;
  start = time(NULL);

  std::string filter_path = "";
  std::string bin_path = "";
  std::string output_path = "";
  switch (argc) {
    case 4:
      parallel_emit_sum = atoi(argv[1]);
      filter_path = argv[2];
      bin_path = argv[3];
      output_path = "origin.txt";
      break;
    case 5:
      parallel_emit_sum = atoi(argv[1]);
      filter_path = argv[2];
      bin_path = argv[3];
      output_path = argv[4];
      break;
    default:
      std::cout << "Please input 3 or 4 paras" << std::endl;
      std::cout << "[parallel emit sum] [filter path] [bin path]" << std::endl;
      std::cout << "[parallel emit sum] [filter path] [bin path] [output path]"
                << std::endl;
      exit(-1);
      break;
  }
  parallel_emit_sum = 16;

  hipError_t cudaStatus;

  time_t start_read, over_read;
  start_read = time(NULL);
  // Read filter data and put in GPU
  ifstream file_read;
  file_read.open(filter_path.c_str(), ios_base::in | ios::binary);
  if (!file_read.is_open()) {
    cout << " the file filter open fail" << endl;
    return -1;
  }
  float filter_data[OD];
  for (int ii = 0; ii < OD; ii++) {
    file_read.read((char *)&filter_data[ii], sizeof(float));
  }
  file_read.close();
  cudaStatus =
    hipMemcpyToSymbol(HIP_SYMBOL(dev_filter_data), filter_data, sizeof(float) * OD);

  if (cudaStatus != hipSuccess) {
    cout << "center Fail to hipMemcpyToSymbol on GPU" << endl;
    return -1;
  }

  file_read.open(bin_path.c_str(), ios_base::in | ios::binary | ios::ate);
  if (!file_read.is_open()) {
    cout << " the bin file open fail" << endl;
    return -1;
  }
  long long int filesize = file_read.tellg() / DEBUG_SAMPLE_RATE_REV;
  file_read.seekg(0, file_read.beg);
  // 为 bin_buffer 申请空间，并把 filepath 的数据载入内存
  char *bin_buffer = (char *)std::malloc(filesize);
  if (bin_buffer == NULL) {
    std::cout << "ERROR :: Malloc data for buffer failed." << std::endl;
    return -1;
  }
  file_read.read(bin_buffer, filesize);
  // if (file_read.peek() == EOF) {
  //   file_read.close();
  // } else {
  //   std::cout << "ERROR :: Read bin file error." << std::endl;
  //   exit(-1);
  // }
  over_read = time(NULL);
  cout << "Reading time is : " << difftime(over_read, start_read) << "s!"
       << endl;

  // image grid
  float ele_coord_x[ELE_NO] = {0};
  float ele_coord_y[ELE_NO] = {0};
  get_ele_position(&ele_coord_x[0], &ele_coord_y[0]);

  if (hipMemcpyToSymbol(HIP_SYMBOL(dev_ele_coord_x), ele_coord_x,
                         sizeof(float) * ELE_NO) != hipSuccess) {
    cout << "ERROR :: Failed for hipMemcpyToSymbol dev_ele_coord_x." << endl;
    return -1;
  }

  if (hipMemcpyToSymbol(HIP_SYMBOL(dev_ele_coord_y), ele_coord_y,
                         sizeof(float) * ELE_NO) != hipSuccess) {
    cout << "ERROR :: Failed for hipMemcpyToSymbol dev_ele_coord_y." << endl;
    return -1;
  }

  float *dev_sumdata;
  int *dev_sumpoint;
  if (hipMalloc((void **)(&dev_sumdata), PIC_RESOLUTION * PIC_RESOLUTION *
                                            sizeof(float)) != hipSuccess) {
    cout << "ERROR :: Failed for hipMalloc dev_sumdata." << endl;
    return -1;
  }
  if (hipMalloc((void **)(&dev_sumpoint), PIC_RESOLUTION * PIC_RESOLUTION *
                                             sizeof(int)) != hipSuccess) {
    cout << "ERROR :: Failed for hipMalloc dev_sumpoint." << endl;
    return -1;
  }
  // init dev_sumdata and dev_sumpoint
  if (hipMemcpy(dev_sumdata, image_data,
                 PIC_RESOLUTION * PIC_RESOLUTION * sizeof(float),
                 hipMemcpyHostToDevice) != hipSuccess) {
    cout << "ERROR :: Failed for hipMemcpy dev_sumdata." << endl;
    return -1;
  }
  if (hipMemcpy(dev_sumpoint, image_point_count,
                 PIC_RESOLUTION * PIC_RESOLUTION * sizeof(int),
                 hipMemcpyHostToDevice) != hipSuccess) {
    cout << "ERROR :: Failed for hipMemcpy dev_sumpoint." << endl;
    return -1;
  }

  long long length_of_data_in_process =
    NSAMPLE * ELE_NO * sizeof(short) * parallel_emit_sum;
  short *dev_data_samples_in_process;
  float *dev_filtered_data;

  cudaStatus = hipMalloc((void **)(&dev_data_samples_in_process),
                          length_of_data_in_process);
  /* if (cudaStatus != hipSuccess)
    {
        cout << "data_samples_in_process Fail to hipMalloc on GPU" << endl;
        return -1;
    } */

  cudaStatus =
    hipMalloc((void **)(&dev_filtered_data), length_of_data_in_process * 2);
  /* if (cudaStatus != hipSuccess) // 转 float 乘以 2
    {
        cout << "ERROR :: Failed for hipMalloc dev_filtered_data." << endl;
        return -1;
    } */

  float *dev_imagedata;

  int *dev_pointcount;

  cudaStatus = hipMalloc((void **)(&dev_imagedata),
                          PIC_RESOLUTION * PIC_RESOLUTION * sizeof(float));
  /* if (cudaStatus != hipSuccess)
    {
        cout << "imagedata Fail to hipMalloc on GPU" << endl;
        //goto Error;
        return cudaStatus;
    } */
  cudaStatus = hipMalloc((void **)(&dev_pointcount),
                          PIC_RESOLUTION * PIC_RESOLUTION * sizeof(int));
  /* if (cudaStatus != hipSuccess)
    {
        cout << "pointcount Fail to hipMalloc on GPU" << endl;
        //goto Error;
        return cudaStatus;
    } */

  long long bin_buffer_index = 0;
  for (int ele_emit_id = 0; ele_emit_id < ELE_NO / DEBUG_SAMPLE_RATE_REV;
       ele_emit_id += parallel_emit_sum) {
    fprintf(stderr, "wavesber of element : %d\n", ele_emit_id);

    // memcpy(&data_samples_in_process[0], &bin_buffer[bin_buffer_index], length_of_data_in_process);
    // bin_buffer_index = bin_buffer_index + length_of_data_in_process;

    // cudaStatus = hipMemcpy(dev_data_samples_in_process,
    // data_samples_in_process, length_of_data_in_process,
    // hipMemcpyHostToDevice);
    cudaStatus =
      hipMemcpy(dev_data_samples_in_process, &bin_buffer[bin_buffer_index],
                 length_of_data_in_process, hipMemcpyHostToDevice);
    bin_buffer_index = bin_buffer_index + length_of_data_in_process;
    if (cudaStatus != hipSuccess) {
      cout << "data_samples_in_process Fail to hipMemcpy on GPU" << endl;
      // goto Error;
      return cudaStatus;
    }
    cudaStatus = precalcWithCuda(
      dev_data_samples_in_process, ele_emit_id, dev_sumdata, dev_sumpoint,
      dev_filtered_data, dev_imagedata, dev_pointcount, parallel_emit_sum);
    //}
    // over=time(NULL);
    // cout<<"Running time is : "<<difftime(over,start)<<"s!"<<endl;
    if (cudaStatus != hipSuccess) {
      fprintf(stderr, "calcWithCuda failed!");
      return 1;
    }
    // hipError_t cudaStatus = calcWithCuda(
    // i,dev_sumdata,dev_sumpoint,dev_filtered_data);
    ::hipDeviceSynchronize();
  }
  cudaStatus = hipMemcpy(image_data, dev_sumdata,
                          PIC_RESOLUTION * PIC_RESOLUTION * sizeof(float),
                          hipMemcpyDeviceToHost);
  /* if (cudaStatus != hipSuccess)
    {
        cout << "allimagedata Fail to hipMemcpy to CPU" << endl;
        return 1;
        //goto Error;
    } */

  cudaStatus = hipMemcpy(image_point_count, dev_sumpoint,
                          PIC_RESOLUTION * PIC_RESOLUTION * sizeof(int),
                          hipMemcpyDeviceToHost);
  /* if (cudaStatus != hipSuccess)
    {
        cout << "allpointcount Fail to hipMemcpy to CPU" << endl;
        return 1;
        //goto Error;
    } */

  write_txtfile(output_path);
  over = time(NULL);
  cout << "Running time is : " << (int)difftime(over, start) / 60 << "min "
       << (int)difftime(over, start) % 60 << "s." << endl;
  hipFree(dev_sumdata);
  hipFree(dev_sumpoint);
  hipFree(dev_data_samples_in_process);
  hipFree(dev_filtered_data);
  hipFree(dev_imagedata);
  hipFree(dev_pointcount);
  // cudaStatus = hipDeviceReset();
  // if (cudaStatus != hipSuccess)
  // {
  //     fprintf(stderr, "hipDeviceReset failed!");
  //     return 1;
  // }
  return 0;
}
